
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__ void cube(float *d_out, float *d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
}

int main(int argc, char **argv){
	const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	float h_in[ARRAY_SIZE];
	float h_out[ARRAY_SIZE];
	for(int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}

	float *d_in, *d_out;
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for(int i = 0; i < ARRAY_SIZE; i++){
		printf("%f", h_out[i]);
		printf("%c", i % 4 == 3? '\n' : '\t');
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}


